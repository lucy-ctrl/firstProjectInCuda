﻿
#include "hip/hip_runtime.h"


#include <stdio.h>


void helloCPU()
{
    printf("Hello from the CPU.\n");
}

__global__ void helloGPU()
{
    printf("Hello from the GPU!\n");
}


int main()
{
    helloCPU();
    helloGPU<<<1, 1>>>();
    hipDeviceSynchronize();

}
